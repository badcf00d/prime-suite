﻿
#include "hip/hip_runtime.h"                                               // CUDA include files...


#include <stdio.h>                                                      // Gives us printf
#include <stdbool.h>                                                    // Gives us the bool type
#include <stdlib.h>                                                     // Gives us dynamic memory functions
#include <math.h>                                                       // Gives us math functions like sqrt


static int* primeList;                                                  // Pointer variable accessible to anything in this file
#define checkCudaError(val) { cudaAssert((val), __FILE__, __LINE__); }  // Macro to provide more useful output on error
#define THREADS_PER_BLOCK 128                                           // Best threads per block from Nvidia Nsight

void cudaAssert(hipError_t code, const char* file, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s)\n", file, line, code, hipGetErrorString(code));
        exit(EXIT_FAILURE);
    }
}


// Factor test by trial division using the 6k +- 1 optimisation, this
// means that factors of factors will not be displayed, i.e. if the test
// number is a factor of 2, it will not show 4, 6, 8 etc.
__global__ void findFactors(int* outputArray, bool verbose)
{
    int const testNum = threadIdx.x + (blockDim.x * blockIdx.x);        // Values set internally that tell us which block & thread this is
    int const testLimit = (int)floor(sqrt((double) testNum));           // Local constant variable
    int* const out = &outputArray[testNum];                             // Local constant pointer to a mutable integer
    bool isPrime = true;

    if (testNum <= 3)
    {
        isPrime = (testNum > 1);
        if (verbose) printf("Special case %d\n", testNum);              // %d means print an integer
    }
    else
    {
        for (int i = 2; i <= 3; i++)                                    // Test for divisibility by 2 and 3
        {
            if ((testNum % i) == 0)
            {
                isPrime = false;
                if (verbose) printf("%d divides by %d\n", testNum, i);
            }
        }
    }

    if (isPrime)
    {
        for (int divisor = 5; divisor <= testLimit; divisor += 6)       // Loop from divisor = 5 to testLimit (inclusive), increment by 6
        {
            if ((testNum % divisor) == 0)                               // Test if it divides by the divisor (i.e. 6k - 1)
            {
                if (verbose) printf("%d divides by %d\n", testNum, divisor);
                isPrime = false;
            }

            if ((testNum % (divisor + 2)) == 0)                         // Test if it divides by the divisor + 2 (i.e. 6k + 1)
            {
                if (verbose) printf("%d divides by %d\n", testNum, divisor + 2);
                isPrime = false;
            }
        }
    }

    if (isPrime)
        *out = testNum;
}


// Helper function for calculating all prime numbers up to maxNumber
static int primeListTest(const int maxNumber)
{
    int const numBlocks = (maxNumber + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK;            // Does a rounded-up division to work out how many blocks of threads we need for the requested number of primes
    int const allocSize = numBlocks * THREADS_PER_BLOCK * sizeof(int);                          // Total number of bytes of memory we need to allocate for the prime number buffers
    int* gpu_primeList;
    int numPrimes = 0;

    primeList = (int*)calloc(allocSize, 1);                                                     // Dynamic memory allocation on the host (i.e. in CPU RAM)

    checkCudaError(hipMalloc((void**)&gpu_primeList, allocSize));                              // Dynamic memory allocation on the GPU
    checkCudaError(hipMemcpy(gpu_primeList, primeList, allocSize, hipMemcpyHostToDevice));    // Copy the contents of primeList (on the host/CPU) to gpu_primeList (on the GPU)

    // Tells the CUDA runtime to make numBlocks blocks each with THREADS_PER_BLOCK threads,
    // e.g if we requested 5 blocks each with 100 threads, the gpu would spawn 500 threads in total.
    // Each thread calls findFactors with the same paramaters. The CUDA runtime sets variables for each
    // thread that we can use to write to the correct memory location from each thread.
    findFactors<<<numBlocks, THREADS_PER_BLOCK>>>(gpu_primeList, false);

    checkCudaError(hipMemcpy(primeList, gpu_primeList, allocSize, hipMemcpyDeviceToHost));    // Copy the contents of gpu_primeList (on the GPU) back to primeList (on the host/CPU)
    checkCudaError(hipFree(gpu_primeList));                                                    // Free the memory on the GPU

    for (int i = 0; i < maxNumber; i++)                                                         // This loop essentially removes the blanks and bunches all the primes up next to eachother in primeList
    {
        if (primeList[i] != 0)
        {
            primeList[numPrimes] = primeList[i];
            numPrimes++;                                                                        // Count up the number of primes we found
        }
    }

    return numPrimes;
}




int main(int argc, char* argv[])
{
    int maxNumber, numPrimes;                                           // local variables only visible to this function
    maxNumber = atoi(argv[1]);                                          // atoi = Ascii TO Integer, argv[0] will be the name of the executable, the first argument is argv[1]
    numPrimes = primeListTest(maxNumber);                               // Calculates all prime numbers up to maxNumber

    printf("Generated %d primes, Largest was: %d \n", numPrimes, primeList[numPrimes - 1]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    checkCudaError(hipDeviceReset());
    hipFree(primeList);

    return 0;
}